#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include <cmath>

#include "core/math.hpp"

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace vllm {

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__device__ __forceinline__ scalar_t compute(const scalar_t& x,
                                            const scalar_t& y) {
  return act_first ? ACT_FN(x) * y : x * ACT_FN(y);
}
// Activation and gating kernel template.

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__global__ void act_and_mul_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = compute<scalar_t, ACT_FN, act_first>(x, y);
  }
}

// NOTE: temporary vectorized version.

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void act_and_mul_kernel_vectorized(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  const int32_t blocks_per_token = gridDim.y;

  const int32_t elems_per_128bit_load = (128 / 8) / sizeof(scalar_t);

  const int32_t tgt_elems_per_block = ceil_div(d, blocks_per_token);
  const int32_t elems_per_block =
      next_multiple_of(elems_per_128bit_load, tgt_elems_per_block);
  const int64_t block_start = blockIdx.y * int64_t(elems_per_block);
  int64_t block_end = block_start + elems_per_block;
  block_end = block_end > d ? d : block_end;

  const scalar_t* __restrict__ x_ptr = input + token_idx * 2 * d;
  const scalar_t* __restrict__ y_ptr = input + token_idx * 2 * d + d;
  scalar_t* __restrict__ out_ptr = out + token_idx * d;

  // 128-bit vectorized code
  const int32_t vec_loop_end =
      prev_multiple_of(elems_per_128bit_load, block_end);
  const int32_t vec_end_idx = vec_loop_end / elems_per_128bit_load;
  const int32_t vec_start_idx = block_start / elems_per_128bit_load;

  const int4* __restrict__ x_128bit_ptr = reinterpret_cast<const int4*>(x_ptr);
  const int4* __restrict__ y_128bit_ptr = reinterpret_cast<const int4*>(y_ptr);
  int4* __restrict__ out_128bit_ptr = reinterpret_cast<int4*>(out_ptr);

#pragma unroll
  for (int32_t vec_idx = vec_start_idx + threadIdx.x; vec_idx < vec_end_idx;
       vec_idx += blockDim.x) {
    const int4 x_128bit = VLLM_LDG(&x_128bit_ptr[vec_idx]);
    const int4 y_128bit = VLLM_LDG(&y_128bit_ptr[vec_idx]);
    using scalar_128bit_vec_t = std::array<scalar_t, elems_per_128bit_load>;

    scalar_128bit_vec_t out_vec;
    const auto x_vec = reinterpret_cast<scalar_128bit_vec_t const&>(x_128bit);
    const auto y_vec = reinterpret_cast<scalar_128bit_vec_t const&>(y_128bit);

#pragma unroll
    for (int i = 0; i < elems_per_128bit_load; i++) {
      out_vec[i] = ACT_FN(x_vec[i]) * y_vec[i];
    }

    out_128bit_ptr[vec_idx] = reinterpret_cast<const int4&>(out_vec);
  }

  // Scalar cleanup code
  if (block_end > vec_loop_end) {
    for (int64_t idx = vec_loop_end + threadIdx.x; idx < block_end;
         idx += blockDim.x) {
      const scalar_t x = VLLM_LDG(&x_ptr[idx]);
      const scalar_t y = VLLM_LDG(&y_ptr[idx]);
      out_ptr[idx] = ACT_FN(x) * y;
    }
  }
}

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <typename T>
__device__ __forceinline__ T gelu_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'none' approximation.
  // Refer to:
  // https://github.com/pytorch/pytorch/blob/8ac9b20d4b090c213799e81acf48a55ea8d437d6/aten/src/ATen/native/cuda/ActivationGeluKernel.cu#L36-L38
  const float f = (float)x;
  constexpr float ALPHA = M_SQRT1_2;
  return (T)(f * 0.5f * (1.0f + ::erf(f * ALPHA)));
}

template <typename T>
__device__ __forceinline__ T gelu_tanh_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'tanh' approximation.
  // Refer to:
  // https://github.com/pytorch/pytorch/blob/8ac9b20d4b090c213799e81acf48a55ea8d437d6/aten/src/ATen/native/cuda/ActivationGeluKernel.cu#L25-L30
  const float f = (float)x;
  constexpr float BETA = M_SQRT2 * M_2_SQRTPI * 0.5f;
  constexpr float KAPPA = 0.044715;
  float x_cube = f * f * f;
  float inner = BETA * (f + KAPPA * x_cube);
  return (T)(0.5f * f * (1.0f + ::tanhf(inner)));
}

}  // namespace vllm

// Launch activation and gating kernel.
// Use ACT_FIRST (bool) indicating whether to apply the activation function
// first.
#define LAUNCH_ACTIVATION_GATE_KERNEL(KERNEL, ACT_FIRST)                 \
  int d = input.size(-1) / 2;                                            \
  int64_t num_tokens = input.numel() / input.size(-1);                   \
  dim3 grid(num_tokens);                                                 \
  dim3 block(std::min(d, 1024));                                         \
  if (num_tokens == 0) {                                                 \
    return;                                                              \
  }                                                                      \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));      \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();          \
  VLLM_DISPATCH_FLOATING_TYPES(                                          \
      input.scalar_type(), "act_and_mul_kernel", [&] {                   \
        vllm::act_and_mul_kernel<scalar_t, KERNEL<scalar_t>, ACT_FIRST>  \
            <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),       \
                                         input.data_ptr<scalar_t>(), d); \
      });

// Launch activation and gating kernel.
// Vectorized Version
#define LAUNCH_ACTIVATION_GATE_KERNEL_VECTORIZED(KERNEL)                 \
  int d = input.size(-1) / 2;                                            \
  int64_t num_tokens = input.numel() / input.size(-1);                   \
  dim3 grid(num_tokens, num_tokens > 16 ? num_tokens > 32 ? 1 : 2 : 4);  \
  dim3 block(std::min(d, 512));                                          \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));      \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();          \
  VLLM_DISPATCH_FLOATING_TYPES(                                          \
      input.scalar_type(), "act_and_mul_kernel_vectorized", [&] {        \
        vllm::act_and_mul_kernel_vectorized<scalar_t, KERNEL<scalar_t>>  \
            <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),       \
                                         input.data_ptr<scalar_t>(), d); \
      });

void silu_and_mul(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL_VECTORIZED(vllm::silu_kernel);
}

void mul_and_silu(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  // The difference between mul_and_silu and silu_and_mul is that mul_and_silu
  // applies the silu to the latter half of the input.
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::silu_kernel, false);
}

void gelu_and_mul(torch::Tensor& out,    // [..., d]
                  torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::gelu_kernel, true);
}

void gelu_tanh_and_mul(torch::Tensor& out,    // [..., d]
                       torch::Tensor& input)  // [..., 2 * d]
{
  LAUNCH_ACTIVATION_GATE_KERNEL(vllm::gelu_tanh_kernel, true);
}

namespace vllm {

template <typename T>
__device__ __forceinline__ T fatrelu_kernel(const T& x, const float threshold) {
  const float f = (float)x;
  return (T)(f > threshold ? f : 0.0f);
}

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&, const float)>
__global__ void act_and_mul_kernel_with_param(
    scalar_t* __restrict__ out, const scalar_t* __restrict__ input, const int d,
    const float param) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = ACT_FN(x, param) * y;
  }
}

}  // namespace vllm

#define LAUNCH_ACTIVATION_GATE_KERNEL_WITH_PARAM(KERNEL, PARAM)         \
  int d = input.size(-1) / 2;                                           \
  int64_t num_tokens = input.numel() / input.size(-1);                  \
  dim3 grid(num_tokens);                                                \
  dim3 block(std::min(d, 1024));                                        \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));     \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();         \
  VLLM_DISPATCH_FLOATING_TYPES(                                         \
      input.scalar_type(), "act_and_mul_kernel_with_param", [&] {       \
        vllm::act_and_mul_kernel_with_param<scalar_t, KERNEL<scalar_t>> \
            <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),      \
                                         input.data_ptr<scalar_t>(), d, \
                                         PARAM);                        \
      });

void fatrelu_and_mul(torch::Tensor& out,    // [..., d],
                     torch::Tensor& input,  // [..., 2 * d]
                     double threshold) {
  LAUNCH_ACTIVATION_GATE_KERNEL_WITH_PARAM(vllm::fatrelu_kernel, threshold);
}
namespace vllm {

// Element-wise activation kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void activation_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * d + idx]);
    out[token_idx * d + idx] = ACT_FN(x);
  }
}

}  // namespace vllm

// Launch element-wise activation kernel.
#define LAUNCH_ACTIVATION_KERNEL(KERNEL)                                       \
  int d = input.size(-1);                                                      \
  int64_t num_tokens = input.numel() / d;                                      \
  dim3 grid(num_tokens);                                                       \
  dim3 block(std::min(d, 1024));                                               \
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));            \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();                \
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "activation_kernel", [&] { \
    vllm::activation_kernel<scalar_t, KERNEL<scalar_t>>                        \
        <<<grid, block, 0, stream>>>(out.data_ptr<scalar_t>(),                 \
                                     input.data_ptr<scalar_t>(), d);           \
  });

namespace vllm {

template <typename T>
__device__ __forceinline__ T gelu_new_kernel(const T& x) {
  const float x3 = (float)(x * x * x);
  const T t = (T)tanhf((T)(0.79788456f * (float)(x + (T)(0.044715f * x3))));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_fast_kernel(const T& x) {
  const float f = (float)x;
  const T t =
      (T)tanhf(((T)(f * 0.79788456f)) * (((T)1.0) + (T)(0.044715f * f) * x));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_quick_kernel(const T& x) {
  // x * sigmoid(1.702 * x)
  return (T)(((float)x) / (1.0f + expf(-1.702f * (float)x)));
}

}  // namespace vllm

void gelu_new(torch::Tensor& out,    // [..., d]
              torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_new_kernel);
}

void gelu_fast(torch::Tensor& out,    // [..., d]
               torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_fast_kernel);
}

void gelu_quick(torch::Tensor& out,    // [..., d]
                torch::Tensor& input)  // [..., d]
{
  LAUNCH_ACTIVATION_KERNEL(vllm::gelu_quick_kernel);
}
